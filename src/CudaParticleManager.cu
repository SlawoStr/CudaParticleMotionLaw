#include "hip/hip_runtime.h"
#include "CudaParticleManager.cuh"
#include ""
#include "RNGGenerator.h"
#include "CudaError.h"
#include <math.h>
#define PI 3.14159265

/// <summary>
/// Calculate signum
/// </summary>
/// <param name="val">Value</param>
/// <returns>Signum of value</returns>
template <typename T> __host__ __device__ int sgn(T val) {
	return (T(0) < val) - (val < T(0));
}

/// <summary>
/// Transfer radians to degrees
/// </summary>
/// <param name="a">Radians</param>
/// <returns>Degrees</returns>
inline __host__ __device__ float degree(float a)
{
	return static_cast<float>(a * (180 / PI));
}

/// <summary>
/// Transfer degree to radians
/// </summary>
/// <param name="a"></param>
/// <returns></returns>
inline __host__ __device__ float radians(float a)
{
	return static_cast<float>(0.017453292 * a);
}

/// <summary>
/// Get distance between two points
/// </summary>
/// <param name="fp">First point</param>
/// <param name="sp">Second point</param>
/// <returns>Distance between points</returns>
__host__ __device__ float getDistance(float2 fp, float2 sp)
{
	return static_cast<float>(sqrtf(powf(sp.x - fp.x, 2) + powf(sp.y - fp.y, 2)));
}

/// <summary>
/// Check if point is on the right side of line
/// </summary>
/// <param name="a">First point of line</param>
/// <param name="b">Second point of line</param>
/// <param name="c">Point</param>
/// <returns>Is point on the right side</returns>
__host__ __device__ bool isRight(float2 a, float2 b, float2 c)
{
	return((b.x - a.x) * (c.y - a.y) - (b.y - a.y) * (c.x - a.x)) > 0;
}

////////////////////////////////////////////////////////////
__global__ void updateAgents(Particles* particles, float particleSpeed, float alpha, float beta, float reactionRadius,float2 simulationBound, int taskSize)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < taskSize; i += blockDim.x * gridDim.x)
	{
		int Nt{};
		int Rt{};
		int Lt{};
		int Ct{};
		// Angle line coordinates
		float2 posStart{ particles[i].position };
		float2 posEnd{ particles[i].position.x + cos(particles[i].angle),particles[i].position.y + sin(particles[i].angle) };
		for (int j = 0; j < taskSize; ++j)
		{
			if (i == j)
				continue;
			float distance{ getDistance(particles[i].position, particles[j].position) };
			if (distance < reactionRadius)
			{
				Nt++;
				if (isRight(posStart, posEnd, particles[j].position))
				{
					Rt++;
				}
				else
				{
					Nt++;
				}
				if (distance < 1.3f)
				{
					Ct++;
				}
			}
		}
		// Update particle data
		float rotationAngle = alpha + beta * Nt * sgn(Rt - Lt);
		particles[i].neighbors = Nt;
		particles[i].closeNeighbors = Ct;
		particles[i].angle += rotationAngle;
		particles[i].nextPosition = { posStart.x + particleSpeed * cos(particles[i].angle),posStart.y + particleSpeed * sin(particles[i].angle) };
		if (particles[i].nextPosition.x < 0.0f || particles[i].nextPosition.x > simulationBound.x || particles[i].nextPosition.y < 0.0f || particles[i].nextPosition.y > simulationBound.y)
		{
			particles[i].nextPosition = particles[i].position;
		}
	}
}

////////////////////////////////////////////////////////////
__global__ void updatePos(Particles* particles,int taskSize)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < taskSize; i += blockDim.x * gridDim.x)
	{
		particles[i].position = particles[i].nextPosition;
	}
}

////////////////////////////////////////////////////////////
CudaParticleManager::CudaParticleManager(float particleSpeed, float alpha, float beta, float reactRadius, float2 simulationBound, int cpuThreadNumber, int threadPerBlock)
	: m_particleSpeed{ particleSpeed }, m_alpha{ radians(alpha) }, m_beta{ radians(beta) }, m_reactionRadius{ reactRadius }, m_simulationBound{ simulationBound },
	m_cpuThreadNumber{ cpuThreadNumber }, m_threadPerBlock{ threadPerBlock }
{
}

////////////////////////////////////////////////////////////
void CudaParticleManager::draw(sf::RenderWindow& window)
{
	sf::CircleShape shape;
	shape.setRadius(1.0f);
	shape.setOrigin(shape.getRadius(), shape.getRadius());

	sf::Vertex line[2];
	line[0].color = sf::Color::Red;
	line[1].color = sf::Color::Red;

	for (const auto& particle : m_cpuParticles)
	{
		shape.setFillColor(sf::Color::Green);
		int n = particle.neighbors;
		if (n > 35)
		{
			shape.setFillColor(sf::Color::Yellow);
		}
		else if (n > 15 && n <= 35)
		{
			shape.setFillColor(sf::Color::Blue);
		}
		else if (n >= 13 && n <= 15)
		{
			shape.setFillColor(sf::Color(136, 69, 19));
		}
		if (particle.closeNeighbors > 15)
		{
			shape.setFillColor(sf::Color::Magenta);
		}
		float2 particlePostion = particle.position;
		shape.setPosition(particlePostion.x, particlePostion.y);
		line[0].position = { particlePostion.x,particlePostion.y };
		line[1].position.x = particlePostion.x + cos(particle.angle);
		line[1].position.y = particlePostion.y + sin(particle.angle);
		window.draw(shape);
		window.draw(line, 2, sf::Lines);
	}
}

////////////////////////////////////////////////////////////
void CudaParticleManager::spawnCells(sf::Vector2f position, int cellNumber)
{
	Particles particle{};
	particle.position = { position.x,position.y };
	// Add new particles to cpu vector and copy it to gpu vector
	for (int i = 0; i < cellNumber; ++i)
	{
		particle.angle = RNGGenerator::randFloat(0.0f, 6.2831f);
		m_cpuParticles.push_back(particle);
	}
	size_t gpuSize = m_gpuParticles.size();
	m_gpuParticles.resize(m_cpuParticles.size());
	checkCudaErrors(hipMemcpy(thrust::raw_pointer_cast(m_gpuParticles.data()) + gpuSize, m_cpuParticles.data() + gpuSize, sizeof(Particles) * cellNumber, hipMemcpyHostToDevice));
}

////////////////////////////////////////////////////////////
void CudaParticleManager::update()
{
	if (m_cpuMode)
	{
		#pragma omp parallel for num_threads(m_cpuThreadNumber)
		for (int i = 0; i < m_cpuParticles.size(); ++i)
		{
			int Nt{};
			int Rt{};
			int Lt{};
			int Ct{};
			// Angle line coordinates
			float2 posStart{ m_cpuParticles[i].position };
			float2 posEnd{ m_cpuParticles[i].position.x + cos(m_cpuParticles[i].angle),m_cpuParticles[i].position.y + sin(m_cpuParticles[i].angle) };
			for (int j = 0; j < m_cpuParticles.size(); ++j)
			{
				if (i == j)
					continue;
				// If particle is in reaction radius
				float distance{ getDistance(m_cpuParticles[i].position, m_cpuParticles[j].position) };
				if (distance < m_reactionRadius)
				{
					Nt++;
					if (isRight(posStart, posEnd, m_cpuParticles[j].position))
					{
						Rt++;
					}
					else
					{
						Nt++;
					}
					if (distance < 1.3f)
					{
						Ct++;
					}
				}
			}
			// Update particle data
			float rotationAngle = m_alpha + m_beta * Nt * sgn(Rt - Lt);
			m_cpuParticles[i].neighbors = Nt;
			m_cpuParticles[i].closeNeighbors = Ct;
			m_cpuParticles[i].angle += rotationAngle;
			m_cpuParticles[i].nextPosition = { posStart.x + m_particleSpeed * cos(m_cpuParticles[i].angle),posStart.y + m_particleSpeed * sin(m_cpuParticles[i].angle) };
			if (m_cpuParticles[i].nextPosition.x < 0.0f || m_cpuParticles[i].nextPosition.x > m_simulationBound.x || m_cpuParticles[i].nextPosition.y < 0.0f || m_cpuParticles[i].nextPosition.y > m_simulationBound.y)
			{
				m_cpuParticles[i].nextPosition = m_cpuParticles[i].position;
			}
		}
		// Update particle positions
		for (int i = 0; i < m_cpuParticles.size(); ++i)
		{
			m_cpuParticles[i].position = m_cpuParticles[i].nextPosition;
		}
	}
	else
	{
		int blockNumber = (static_cast<int>(m_gpuParticles.size()) + m_threadPerBlock - 1) / m_threadPerBlock;
		updateAgents << <blockNumber, m_threadPerBlock >> > (thrust::raw_pointer_cast(m_gpuParticles.data()), m_particleSpeed, m_alpha, m_beta, m_reactionRadius, m_simulationBound, static_cast<int>(m_gpuParticles.size()));
		updatePos << <blockNumber, m_threadPerBlock >> > (thrust::raw_pointer_cast(m_gpuParticles.data()), static_cast<int>(m_gpuParticles.size()));
		checkCudaErrors(hipMemcpy(m_cpuParticles.data(), thrust::raw_pointer_cast(m_gpuParticles.data()), sizeof(Particles) * m_cpuParticles.size(), hipMemcpyDeviceToHost));
	}
}

////////////////////////////////////////////////////////////
void CudaParticleManager::reduceSimulationBound()
{
	m_simulationBound.x -= 1;
	m_simulationBound.y -= 1;

	for (int i = 0; i < m_cpuParticles.size(); ++i)
	{
		if (m_cpuParticles[i].position.x >= m_simulationBound.x)
		{
			m_cpuParticles[i].position.x -= 1;
		}
		if (m_cpuParticles[i].position.y >= m_simulationBound.y)
		{
			m_cpuParticles[i].position.y -= 1;
		}
	}
	checkCudaErrors(hipMemcpy(thrust::raw_pointer_cast(m_gpuParticles.data()), m_cpuParticles.data(), sizeof(Particles) * m_cpuParticles.size(), hipMemcpyHostToDevice));
}

////////////////////////////////////////////////////////////
void CudaParticleManager::increaseSimulationBound()
{
	m_simulationBound.x += 1;
	m_simulationBound.y += 1;
}

////////////////////////////////////////////////////////////
float2 CudaParticleManager::getSimulationBound() const
{
	return m_simulationBound;
}

////////////////////////////////////////////////////////////
void CudaParticleManager::changeMode()
{
	if (m_cpuMode == true)
	{
		checkCudaErrors(hipMemcpy(thrust::raw_pointer_cast(m_gpuParticles.data()), m_cpuParticles.data(), sizeof(Particles) * m_cpuParticles.size(), hipMemcpyHostToDevice));
	}
	else
	{
		checkCudaErrors(hipMemcpy(m_cpuParticles.data(), thrust::raw_pointer_cast(m_gpuParticles.data()), sizeof(Particles) * m_cpuParticles.size(), hipMemcpyDeviceToHost));
	}
	m_cpuMode = !m_cpuMode;
}
